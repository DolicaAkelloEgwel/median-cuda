
#include <hip/hip_runtime.h>
extern "C"{
    __device__ float find_median_in_one_dim_array(float* neighb_array, const int N)
    {
        int i, j;
        float key;

        for (i = 1; i < N; i++)
        {
            key = neighb_array[i];
            j = i - 1;

            while (j >= 0 && neighb_array[j] > key)
            {
                neighb_array[j + 1] = neighb_array[j];
                j = j - 1;
            }
            neighb_array[j + 1] = key;
        }
        return neighb_array[N / 2];
    }
    __device__ void print_neighbour_array(const float* array, const int N)
    {
        for (int i = 0; i < N; i++)
            printf("%.6f ", array[i]);
        printf("\n");
    }
    __device__ float find_neighbour_median(const float* padded_array, const int index_offset, const int padded_img_width, const int id_x, const int id_y, const int filter_size)
    {
        float neighb_array[25];
        int n_counter = 0;

        for (int i = id_x; i < id_x + filter_size; i++)
        {
            for (int j = id_y; j < id_y + filter_size; j++)
            {
                neighb_array[n_counter] = padded_array[index_offset + (i * padded_img_width) + j];
                n_counter += 1;
            }
        }

        return find_median_in_one_dim_array(neighb_array, filter_size * filter_size);
    }
    __global__ void three_dim_median_filter(float* data_array, const float* padded_array, const int N_IMAGES, const int X, const int Y, const int filter_size)
    {
        unsigned int id_img = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int id_x = blockIdx.y*blockDim.y + threadIdx.y;
        unsigned int id_y = blockIdx.z*blockDim.z + threadIdx.z;
        unsigned int img_size =  X * Y;
        unsigned int padded_img_width =  X + filter_size - 1;
        unsigned int padded_img_size =  padded_img_width * (Y + filter_size - 1);

        if ((id_img >= N_IMAGES) || (id_x >= X) || (id_y >= Y))
            return;

        data_array[(id_img * img_size) + (id_x * X) + id_y] = find_neighbour_median(padded_array, id_img * padded_img_size, padded_img_width, id_x, id_y, filter_size);
    }
    __global__ void two_dim_median_filter(float* data_array, const float* padded_array, const int X, const int Y, const int filter_size)
    {
        unsigned int id_x = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int id_y = blockIdx.y*blockDim.y + threadIdx.y;
        unsigned int padded_img_width =  X + filter_size - 1;

        if ((id_x >= X) || (id_y >= Y))
            return;

        data_array[(id_x * X) + id_y] = find_neighbour_median(padded_array, 0, padded_img_width, id_x, id_y, filter_size);  
    }
    __global__ void two_dim_remove_light_outliers(float* data_array, const float* padded_array, const int X, const int Y, const int filter_size, const float diff)
    {
        unsigned int id_x = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int id_y = blockIdx.y*blockDim.y + threadIdx.y;
        unsigned int index = (id_x * X) + id_y;
        unsigned int padded_img_width =  X + filter_size - 1;

        if ((id_x >= X) || (id_y >= Y))
            return;

        float median = find_neighbour_median(padded_array, 0, padded_img_width, id_x, id_y, filter_size);

        if (data_array[index] - median >= diff)
            data_array[index] = median;
    }
    __global__ void two_dim_remove_dark_outliers(float* data_array, const float* padded_array, const int X, const int Y, const int filter_size, const float diff)
    {
        unsigned int id_x = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int id_y = blockIdx.y*blockDim.y + threadIdx.y;
        unsigned int index = (id_x * X) + id_y;
        unsigned int padded_img_width =  X + filter_size - 1;

        if ((id_x >= X) || (id_y >= Y))
            return;

        float median = find_neighbour_median(padded_array, 0, padded_img_width, id_x, id_y, filter_size);

        if (median - data_array[index] >= diff)
            data_array[index] = median;
    }
}
